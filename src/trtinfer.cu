#include "hip/hip_runtime.h"
#include <string>
#include <fstream>
#include <iostream>
#include "trtinfer.cuh"
#include <chrono>
#include "timer.hpp"
#include "NvOnnxParser.h"
#include <memory>
#include <string>

auto sizer = [](nvinfer1::Dims dims){size_t product = 1; for(int i=0 ; i<dims.nbDims ; i++)	product*=dims.d[i];	return product;};		//Helper Lambda

template <typename T>
bool TRTInfer<T>::loadTRTEngine(std::string engine_filename, int DLACore)
{
	std::ifstream engineFile(engine_filename, std::ios::binary);
    if (!engineFile)	{std::cout << "\nError opening engine file: ";	return false;}

    engineFile.seekg(0, engineFile.end);
    long int fsize = engineFile.tellg();
    engineFile.seekg(0, engineFile.beg);

    std::vector<char> engineData(fsize);
    engineFile.read(engineData.data(), fsize);
	engineFile.close();
    params.runtime = nvinfer1::createInferRuntime(params.Logger);

	if(DLACore>=0)	params.runtime->setDLACore(DLACore);	//Have to do this coz for some reason getDLAcore returns -1 when using trtexec generated files, seems like an API bug

    params.engine = params.runtime->deserializeCudaEngine(engineData.data(), fsize); 
    params.context = params.engine->createExecutionContext();
	
	for(int i=0 ; i<params.engine->getNbBindings() ; i++)	
		if(params.engine->bindingIsInput(i))	Ninput_tensors++ ;
		else									Noutput_tensors++;
	
    engineFile.close();
	return true;
}

template <typename T>
void TRTInfer<T>::allocITensors()
{
	input_tensors.resize(Ninput_tensors);
	for(int i=0 ; i<input_tensors.size() ; i++)	
	{
		input_tensors[i].shape = params.engine->getBindingDimensions(i);
		Ialloc = input_tensors[i].allocTensor();	
	}	
}
//
template <typename T>
void TRTInfer<T>::allocOTensors()
{
	output_tensors.resize(Noutput_tensors);
	for(int i=0 ; i<output_tensors.size() ; i++)	
	{
		output_tensors[i].shape = params.engine->getBindingDimensions(i + Ninput_tensors);
		Oalloc = output_tensors[i].allocTensor();	
	}
}
//
template <typename T>
void TRTInfer<T>::allocIOTensors()
{
	allocITensors();
	allocOTensors();
}
//
template <typename T>
void TRTInfer<T>::shallowcopyITensors(TRTInfer<T> &ITensorCarryingObj)
{
	for (auto &i:ITensorCarryingObj.output_tensors)	input_tensors.push_back(i);
}
//
template <typename T>
void TRTInfer<T>::infer(hipStream_t &stream)
{
	std::vector<void *> buffers;
	
	for(int i=0 ; i<input_tensors.size() ; i++)		buffers.push_back(input_tensors[i].data);
	for(int i=0 ; i<output_tensors.size() ; i++)	buffers.push_back(output_tensors[i].data);
	
	bool success = params.context->enqueueV2(buffers.data(), stream, nullptr);
	if (!success)
	{
       std::cerr << "Inference Error in " << __FILE__ << " at line " << __LINE__ << '\n';
       exit(1);
	}
}
//
template class TRTInfer<float>;
template class TRTInfer<__half>;
//template class TRTInfer<double>;
//
//template class TRTInfer<int>;
//template class TRTInfer<unsigned int>;
//
//template class TRTInfer<long>;
//template class TRTInfer<unsigned long>;
//
//template class TRTInfer<short>;
//template class TRTInfer<unsigned short>;
//
//template class TRTInfer<char>;
//template class TRTInfer<unsigned char>;

template <typename T>
[[nodiscard]] bool tensor<T>::allocTensor()	
{
	if(shapeCorrectness())	return (!(hipMalloc(&data, sizer(shape) * sizeof(T))));	
	return false;
}

template <typename T>
[[nodiscard]] bool tensor<T>::freeTensor()	
{
	if(data!=nullptr)	return (!hipFree(data));	
	return false;
}

template <typename T>
[[nodiscard]] bool tensor<T>::shapeCorrectness()
{
	if (shape.nbDims && sizer(shape)>0)	return true;
	return false;
}
