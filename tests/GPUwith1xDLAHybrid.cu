#include "DLATestCommon.cuh"
#include "GPUTestCommon.cuh"

void GPUwithHybridTest(std::string filepath_GPUFull, std::string filepath_G1, std::string filepath_G2)
{
	std::thread GPU(GPUOnly, filepath_GPUFull);
	std::thread DLACore0(G1G2FullHybridInfer, 0, filepath_G1, filepath_G2);
	
	GPU.join();
	DLACore0.join();
}

int main()
{	
	GPUwithHybridTest(file_locations::FullModel_filepath, file_locations::G1_filepath, file_locations::G2_filepath);
}
