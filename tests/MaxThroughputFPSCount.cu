#include "DLATestCommon.cuh"
#include "GPUTestCommon.cuh"

bool countstart = false;
bool firststart = true;
bool breakDLA = false;
bool dlafirststart[] = {false, false};

unsigned long localEPOCH;
unsigned long gpuFrames=0, dlaFrames[] = {0, 0};

void framecountGPU(std::string filepath_GPUFull, long warmup_time_ms, long infer_time_ms)
{
	if(warmup_time_ms<=0 || infer_time_ms<=0)	{std::cout << "\nInvalid Run duration recieved!!"; return;}
	long counter = 1;
	TRTInfer<> GPU;
	GPU.loadTRTEngine(filepath_GPUFull);
	GPU.allocIOTensors();
	hipStream_t gpustream;
	hipStreamCreate (&gpustream);
	HresTimer T("GPU");
	while (++counter)
	{
		if(((std::chrono::system_clock::now().time_since_epoch() / std::chrono::milliseconds(1))-localEPOCH) >= warmup_time_ms)	{	countstart = true;	}
		if(countstart && firststart) {	T.start_timer();	firststart=false;	counter = 1;	printf("\nTEST STARTED GPU");	dlafirststart[0] = true;	dlafirststart[1] = true;}
		
		GPU.infer(gpustream);
		hipStreamSynchronize(gpustream);
		
		if(((std::chrono::system_clock::now().time_since_epoch() / std::chrono::milliseconds(1))-localEPOCH) >= warmup_time_ms + infer_time_ms) {	breakDLA = true;	break;	}
	}
	T.stop_timer(1);
	gpuFrames = counter;
	printf("\nGPU FRAMES : %ld", counter);
	printf("\nGPU FPS = %f\n",  1000*(float)counter/infer_time_ms);
}

void framecountDLA(std::string filepath_G1, std::string filepath_G2, int core, long warmup_time_ms, long infer_time_ms)
{
	long counter = 1;
	
	TRTInfer<> DLAG1;
	DLAG1.loadTRTEngine(filepath_G1, core);
	DLAG1.allocIOTensors();
	
	TRTInfer<> G2;
	G2.loadTRTEngine(filepath_G2);
	G2.allocOTensors();
		
	hipStream_t dlastream, gpustream;
	hipStreamCreate (&dlastream);
	hipStreamCreate (&gpustream);
	
	HresTimer T("Hybrid Infer");
	while (!breakDLA)
	{
		counter++;
		if(countstart && dlafirststart[core] )	{	counter = 1;	printf("\nTEST STARTED DLA %d ACK", core);	T.start_timer();	dlafirststart[core] = false;}
		
		DLAG1.infer(dlastream);
		hipStreamSynchronize(dlastream);
		
		G2.shallowcopyITensors(DLAG1);
		G2.infer(gpustream);
		hipStreamSynchronize(gpustream);
	}
	T.stop_timer(1);
	dlaFrames[core] = counter;
	printf("\nDLA FRAMES Core %d : %ld", core, counter);
	printf("\nDLA FPS = %f\n",  1000*(float)counter/infer_time_ms);
}

void framecountTEST(std::string filepath_GPUFull, std::string filepath_G1, std::string filepath_G2, long warmup_time_ms = 30000 , long infer_time_ms = 100000)
{
	printf("\nTest Warmup = %ld ms\nTest Infer = %ld ms\n" , warmup_time_ms, infer_time_ms);
	::localEPOCH = std::chrono::system_clock::now().time_since_epoch() / std::chrono::milliseconds(1);
	
	std::thread GPUFrameTest(framecountGPU, filepath_GPUFull, warmup_time_ms, infer_time_ms);
	
	std::thread HybridTestDLA1(framecountDLA, filepath_G1, filepath_G2, 0, warmup_time_ms, infer_time_ms);
	std::thread HybridTestDLA2(framecountDLA, filepath_G1, filepath_G2, 1, warmup_time_ms, infer_time_ms);
	
	GPUFrameTest.join();
	HybridTestDLA1.join();
	HybridTestDLA2.join();
	
	printf("\nTotal Frames Processed = %lu" , (gpuFrames + dlaFrames[0] + dlaFrames[1]));
	printf("\nThroughput : %f FPS\n", float(1000*(gpuFrames + dlaFrames[0] + dlaFrames[1]))/infer_time_ms);
}

int main()
{		
	//Set warmup to atleast 30000ms or more for ideal behaviour
	
	size_t warmup_duration_ms = 30000, infer_duration_ms = 30000;
	framecountTEST(file_locations::FullModel_filepath, file_locations::G1_filepath, file_locations::G2_filepath, warmup_duration_ms, infer_duration_ms);
}