#include "GPUTestCommon.cuh"

void GPUTest(std::string filepath_GPUFull)
{
	std::thread GPU(GPUOnly, filepath_GPUFull);
	
	GPU.join();
}


int main()
{	
	GPUTest(file_locations::FullModel_filepath);
}
