#include "DLATestCommon.cuh"

void HybridTest(std::string filepath_G1, std::string filepath_G2)
{
	std::thread DLACore0(G1G2FullHybridInfer, 0, filepath_G1, filepath_G2);
	std::thread DLACore1(G1G2FullHybridInfer, 1, filepath_G1, filepath_G2);
	
	DLACore0.join();
	DLACore1.join();
}

int main()
{	
	HybridTest(file_locations::G1_filepath, file_locations::G2_filepath);
}
